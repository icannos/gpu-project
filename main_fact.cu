#include "hip/hip_runtime.h"
//
// Pierre G 27/03/20.
//


// Usage
// ./build/fact N d num_thread_per_block
// ./build/fact N d num_thread_per_block  |  python verify_facto.py --atol 0.01


#include "LDLt.h"

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <cmath>


int main(int argc, char* argv[]) {
    float Tim;                            // GPU timer instructions
    hipEvent_t start, stop;            // GPU timer instructions
    int d = 20;
    int n = 5;
    int num_thread_per_block = 1024;
    int factorizer = 0; // 0: columns || 1: rows || 2: shared memory+row

    n = atoi(argv[1]);
    d = atoi(argv[2]);
    num_thread_per_block = atoi(argv[3]);
    if (argc>4)
      factorizer = atoi(argv[4]);

    // int minTB = 1;  // number of matrix per block
    int minTB = num_thread_per_block/d;  // number of matrix per block
    int NB = (n+minTB-1)/minTB;  // number of blocks (round up)
    printf("%d %d", minTB, NB);

    srand(time(0));

    auto *A     = (float *) malloc(sizeof(float) * n * (d + d * (d + 1) / 2));
    auto *LandD = (float *) malloc(sizeof(float) * n * (d + d * (d + 1) / 2));
    auto *Y = (float *) malloc(sizeof(float) * n * d);
    // auto *Ychap = (float *) malloc(sizeof(float) * n * d);
    auto *X = (float *) malloc(sizeof(float) * n * d);

    float *gpuA;
    float *gpuY;

    hipMalloc(&gpuA, sizeof(float) * n * (d + d * (d + 1) / 2));
    hipMalloc(&gpuY, sizeof(float) * n * d);

    generate_systems(A, Y, n, d, false);

    hipMemcpy(gpuA, A, sizeof(float) * n * (d + d * (d + 1) / 2), hipMemcpyHostToDevice);
    // hipMemcpy(gpuY, Y, sizeof(float) * n * d, hipMemcpyHostToDevice);



    hipEventCreate(&start);                // GPU timer instructions
    hipEventCreate(&stop);                 // GPU timer instructions
    hipEventRecord(start, 0);              // GPU timer instructions

    // LDLt_max_col_k <<< NB, d * minTB, minTB * ((d * d + d) / 2 + d) * sizeof(float) >>> (gpuA, d);
    // LDLt_max_row_k <<< NB, d * minTB, minTB * ((d * d + d) / 2 + d) * sizeof(float) >>> (gpuA, d);
    if (factorizer==0)
        LDLt_max_col_k <<< NB, d * minTB, 0 >>> (gpuA, d);
    else if (factorizer==1)
        LDLt_max_row_k <<< NB, d * minTB, 0 >>> (gpuA, d);
    else if (factorizer==2)
        LDLt_max_row_k_SHARED <<< NB, d * minTB, minTB * ((d * d + d) / 2 + d) * sizeof(float) >>> (gpuA, d);
    else
        throw std::invalid_argument( "unknown factorizer" );
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);               // GPU timer instructions
    hipEventSynchronize(stop);             // GPU timer instructions
    hipEventElapsedTime(&Tim, start, stop);// GPU timer instructions
    hipEventDestroy(start);                // GPU timer instructions
    hipEventDestroy(stop);                 // GPU timer instructions
    printf("\nExecution time %f ms\n", Tim);  // GPU timer instructions


    hipMemcpy(LandD, gpuA, sizeof(float) * n * (d + d * (d + 1) / 2), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // hipMemcpy(X, gpuY, sizeof(float) * n * d, hipMemcpyDeviceToHost);
    // hipDeviceSynchronize();



    // printf("[");
    // for(int k = 0; k<d; k++)
    //     printf("%f,", X[k]);
    // printf("]\n");
    // printf("\0");

    int A_size = d + d * (d + 1) / 2;
    printf("{\n");
    // print A
    printf("'A':[");
    for (int midx=0; midx<n; midx++){
      printf("[");
      for(int i = 0; i<d; i++){
        printf("[");
        for(int j = 0; j<d; j++){
          if(i==j)
            printf("%f,",A[midx*A_size+hgetDPierre(i)]);
          else
            printf("%f,",A[midx*A_size+d+hgetLPierre(d,i,j)]);
        }
        printf("],");
      }
      printf("],");
    }
    printf("],\n");
    // printf("\0");

    // print L
    printf("'L':[");
    for (int midx=0; midx<n; midx++){
      printf("[");
      for(int i = 0; i<d; i++){
        printf("[");
        for(int j = 0; j<d; j++){
          if(j>i)
            printf("%f,",0.0f);
          else
            printf("%f,",LandD[midx*A_size+d+hgetLPierre(d,i,j)]);
        }
        printf("],");
      }
      printf("],");
    }
    printf("],\n");
    // printf("\0");

    // print D
    printf("'D':[");
    for (int midx=0; midx<n; midx++){
      printf("[");
      for(int i = 0; i<d; i++)
        printf("%f,",LandD[midx*A_size+hgetDPierre(i)]);
      printf("],");
    }
    printf("],\n");
    // printf("\0");

    // print ones
    printf("'ones':[");
    for (int midx=0; midx<n; midx++){
      printf("[");
      for(int i = 0; i<d; i++)
        printf("%f,",A[midx*A_size+d+hgetLPierre(d, i, i)]);
      printf("],");
    }
    printf("],\n");
    // printf("\0");

    printf("}\n");


    hipFree(gpuA);
    hipFree(gpuY);
    free(A);
    free(LandD);
    free(Y);
    // free(Ychap);
    // free(X);

    return 0;
}
