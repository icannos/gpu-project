

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <ctime>

int hgetLPierre(int d, int i, int j) {
    // If j > i, then we take the transpose of L
    if (j > i) {
        int t = i;
        i = j;
        j = t;
    }

    int l_position    =     i*(i+1) / 2 + j;
    // int l_position =     i*(i-1) / 2 + j-1;

    return l_position;
}

int hgetDPierre(int i) {
    int d_position = i;
    return d_position;
}

void generate_systems(float *A, float *Y, int N, int d, bool verbose=true) {
    int matrix_size = d + d * (d + 1) / 2;

    for (int i = 0; i < N; i++) {
        float *D = &A[i * matrix_size];
        float *T = &A[i * matrix_size + d];


        for (int j=0; j < (d * (d + 1) / 2); j++)
            T[j] = ((float) rand()+1)*1./RAND_MAX;


        for (int j=0; j<d; j++)
            T[hgetLPierre(d, j,j)] = 1.0f;

        for (int j=0; j<d; j++) {
            D[j] = ((float) rand()+1)*1./RAND_MAX;
            Y[d * i + j] = ((float) rand()+1)*1./RAND_MAX;
        }

    }
}

// ************************************************************************ //

// __device__ int getLPierre(float* T, int n, int d, int matrix_id, int i, int j)
__device__ int getLPierre(int d, int i, int j)
{
    // If j > i, then we take the transpose of L
    if (j > i) {int t = i; i = j; j = t;};

    // int matrix_memory_size = (d+d*(d+1)/2);
    int l_position    = d + i*(i+1) / 2 + j;
    // int l_position =     i*(i-1) / 2 + j-1;
    // int l_position = d + i*(i-1) / 2 + j;
    return l_position;
    // return &T[matrix_id * matrix_memory_size + l_position]
}

// __device__ int getDPierre(float* T, int n, int d, int matrix_id, int i)
__device__ int getDPierre(int d, int i)
{
    // int matrix_memory_size = (d+d*(d+1)/2);
    int d_position = i;
    return d_position;
    // return &T[matrix_id * matrix_memory_size + d_position]
}

__device__ void parallel_copy(float* src, float* dest, int n)
{
    int i = threadIdx.x;
    int stride = blockDim.x;
    while(i<n){
        dest[i] = src[i];
        i += stride;
    }
    __syncthreads();
}

// __global__ void LDLt_max_col_k(float* AGPU, int d)
__global__ void LDLt_max_col_k(float* sA, int d)
{
    int tidx = threadIdx.x%d;
    int Qt = (threadIdx.x-tidx)/d;
    int A_size = d*(d+1)/2+d;
    int minTB = blockDim.x/d;
    // printf("minTB %d\n", minTB);
    int nt = (blockIdx.x*minTB + Qt) * A_size;
    // int gbx = Qt + blockIdx.x*(blockDim.x/d);


    // extern __shared__ float sA[];
    // //copy ACPU to sA
    // parallel_copy(sA, &AGPU[(blockIdx.x*minTB + Qt)*A_size], minTB*A_size);

    // tidx==i

    // Perform the LDLt factorization
    int j, k;
    for(j=0; j<d; j++){
        // D_j,j :
        if(tidx==0){
            for(k=0; k<j; k++){
                sA[nt+getDPierre(d, j)] -= sA[nt+getDPierre(d,k)]*
                                     sA[nt+getLPierre(d,j,k)]*
                                     sA[nt+getLPierre(d,j,k)];
            }
        }
        __syncthreads();

        // L_:,j parallel
        if(tidx>j){
            //printf("(%d,%d,%d,%d),", nt+getLPierre(d,tidx,j), nt, tidx, j);
            sA[nt+getLPierre(d,tidx,j)] /= sA[nt+getDPierre(d,j)];
            for(k=0; k<j; k++){
                sA[nt+getLPierre(d,tidx,j)] -= sA[nt+getLPierre(d,tidx,k)]*
                                         sA[nt+getLPierre(d,j,k)]*
                                         sA[nt+getDPierre(d,k)]/
                                         sA[nt+getDPierre(d,j)];
            }
        }
        __syncthreads();
    }

    // parallel_copy(&AGPU[(blockIdx.x*minTB + Qt)*A_size], sA, minTB*A_size);

}

// __global__ void LDLt_max_row_k(float* AGPU, int d)
__global__ void LDLt_max_row_k(float* sA, int d)
{
    int tidx = threadIdx.x%d;
    int Qt = (threadIdx.x-tidx)/d;
    int A_size = d*(d+1)/2+d;
    int minTB = blockDim.x/d;
    // printf("minTB %d\n", minTB);
    int nt = (blockIdx.x*minTB + Qt) * A_size;
    // int gbx = Qt + blockIdx.x*(blockDim.x/d);


    // extern __shared__ float sA[];
    // //copy ACPU to sA
    // parallel_copy(sA, &AGPU[(blockIdx.x*minTB + Qt)*A_size], minTB*A_size);
    // Perform the LDLt factorization
    int i, k;
    for(i=0; i<d; i++){
        // D_i,i :
        if(tidx==0){
            for(k=0; k<i; k++){
                sA[nt+getDPierre(d, i)] -= sA[nt+getDPierre(d,k)]*
                                     sA[nt+getLPierre(d,i,k)]*
                                     sA[nt+getLPierre(d,i,k)];
            }
        }
        __syncthreads();

        // L_i,: parallel
        if(i<tidx){
            //printf("(%d,%d,%d,%d),", nt+getLPierre(d,i,tidx), nt, i,tidx);
            sA[nt+getLPierre(d,i,tidx)] /= sA[nt+getDPierre(d,i)];
            for(k=0; k<i; k++){
                sA[nt+getLPierre(d,i,tidx)] -= sA[nt+getLPierre(d,k,tidx)]*
                                         sA[nt+getLPierre(d,k,i)]*
                                         sA[nt+getDPierre(d,k)]/
                                         sA[nt+getDPierre(d,i)];
            }
        }
        __syncthreads();
    }

    // parallel_copy(&sA[(blockIdx.x*minTB + Qt)*A_size], A_host, minTB*A_size);
}

// __global__ void LDLt_max_row_k(float* AGPU, int d)
__global__ void LDLt_max_row_k_SHARED(float* A_host, int d)
{
    int tidx = threadIdx.x%d;
    int Qt = (threadIdx.x-tidx)/d;
    int A_size = d*(d+1)/2+d;
    int minTB = blockDim.x/d;
    // printf("minTB %d\n", minTB);
    int nt = Qt * A_size;
    // int gbx = Qt + blockIdx.x*(blockDim.x/d);

    extern __shared__ float sA[];
    //copy ACPU to sA
    parallel_copy(&A_host[blockIdx.x*minTB*A_size], sA, minTB*A_size);

    // tidx==j

    // Perform the LDLt factorization
    int i, k;
    for(i=0; i<d; i++){
        // D_i,i :
        if(tidx==0){
            for(k=0; k<i; k++){
                sA[nt+getDPierre(d, i)] -= sA[nt+getDPierre(d,k)]*
                                     sA[nt+getLPierre(d,i,k)]*
                                     sA[nt+getLPierre(d,i,k)];
            }
        }
        __syncthreads();

        // L_i,: parallel
        if(i<tidx){
            //printf("(%d,%d,%d,%d),", nt+getLPierre(d,i,tidx), nt, i,tidx);
            sA[nt+getLPierre(d,i,tidx)] /= sA[nt+getDPierre(d,i)];
            for(k=0; k<i; k++){
                sA[nt+getLPierre(d,i,tidx)] -= sA[nt+getLPierre(d,k,tidx)]*
                                         sA[nt+getLPierre(d,k,i)]*
                                         sA[nt+getDPierre(d,k)]/
                                         sA[nt+getDPierre(d,i)];
            }
        }
        __syncthreads();
    }

    parallel_copy(sA, &A_host[blockIdx.x*minTB*A_size], minTB*A_size);
}


// ************************************************************************ //
