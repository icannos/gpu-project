#include "hip/hip_runtime.h"
/**************************************************************
Pierre Guetschel

***************************************************************/

#include <iostream>


// Genrates Gaussian distribution from a uniform one (Box-Muller)
__device__ void BoxMuller_d(float *g0, float *g1) {

    float loc;
    if (*g1 < 1.45e-6f) {
        loc = sqrtf(-2.0f * logf(0.00001f)) * cosf(*g0 * 2.0f * MoPI);
    } else {
        if (*g1 > 0.99999f) {
            loc = 0.0f;
        } else { loc = sqrtf(-2.0f * logf(*g1)) * cosf(*g0 * 2.0f * MoPI); }
    }
    *g0 = loc;
}

// Monte Carlo routine
__global__ void LDLt_max_k(int AGPU, int YGPU, int d) {
    int tidx = threadIdx.x % d;
    int Qt = (threadIdx.x - tidx) / d;
    int gbx = Qt + blockIdx.x * (blockDim.x / d);


    extern __shared__ float H[];

// Perform the LDLt factorization
    for (i = n; i > 0; i--) {
        if (tidx == 0) {
            for (k = n; k > i; k--) {
                sA[nt + n2 - i * (i + 1) / 2] -= sA[nt + n2 - k * (k + 1) / 2] *
                                                 sA[nt + n2 - k * (k + 1) / 2 + k - i] *
                                                 sA[nt + n2 - k * (k + 1) / 2 + k - i];
            }
        }
        __syncthreads();
        if (tidx < i - 1) {
            sA[nt + n2 - i * (i + 1) / 2 + tidx + 1] /= sA[nt + n2 - i * (i + 1) / 2];
            for (k = n; k > i; k--) {
                sA[nt + n2 - i * (i + 1) / 2 + tidx + 1] -= sA[nt + n2 - k * (k + 1) / 2] *
                                                            sA[nt + n2 - k * (k + 1) / 2 + k - i] *
                                                            sA[nt + n2 - k * (k + 1) / 2 + tidx + 1 + k - i] /
                                                            sA[nt + n2 - i * (i + 1) / 2];
            }
        }
        __syncthreads();
    }

}


int main() {
    float Tim;                            // GPU timer instructions
    hipEvent_t start, stop;            // GPU timer instructions

    hipMalloc(&res1, sizeof(float));
    hipMemset(res2, 0.0f, sizeof(float));


    hipEventCreate(&start);            // GPU timer instructions
    hipEventCreate(&stop);                // GPU timer instructions
    hipEventRecord(start, 0);            // GPU timer instructions

    LDLt_max_k << < NB, d * minTB, minTB * ((d * d + d) / 2 + d) * sizeof(float) >> > (AGPU, YGPU, d);

    hipEventRecord(stop, 0);            // GPU timer instructions
    hipEventSynchronize(stop);            // GPU timer instructions
    hipEventElapsedTime(&Tim,            // GPU timer instructions
                         start, stop);                // GPU timer instructions
    hipEventDestroy(start);            // GPU timer instructions
    hipEventDestroy(stop);                // GPU timer instructions


    hipMemcpy(&sum, res1, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(res1);


    printf("Execution time %f ms\n", Tim);

    return 0;
}
