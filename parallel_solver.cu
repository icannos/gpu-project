#include "hip/hip_runtime.h"
//
// Created by maxime on 08/03/20.
//

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include "ide_params.h"
#include "parallel_solver.h"


__device__ int getL(int d, int i, int j) {
    // If j > i, then we take the transpose of L
    if (j > i) {
        int t = i;
        i = j;
        j = t;
    }

    int l_position = i * (i - 1) / 2 + j - 1;

    return l_position;
}

__device__ int getD(int i) {
    int d_position = i;
    return d_position;
}

__device__ void reduce_sum(float *T, int size) {
    __syncthreads();
    if (size > 1) {
        for (unsigned int s = 1; s < size; s *= 2) {
            __syncthreads();
            if (threadIdx.x % (2 * s) == 0) {
                T[threadIdx.x] += T[threadIdx.x + s];
            }
        }
    }
}

__device__ void solve_tinf(float *T, float *Y, int d) {
    // Solve an equation of the form LZ = Y
    // T represents the inf triangular matrix
    // The results is then stored in Z

    extern __shared__ float tmp[];

    int blockdim = blockDim.x;
    int threadid = threadIdx.x;

    for (int i = 1; i < d; i++) {
        int q = i / blockdim;
        int rmd = i % blockdim;

        // general case
        __syncthreads();
        for (int k = 0; k < q; k++) {


            if (threadid < blockdim) {

                tmp[threadid] = Y[k * blockdim + threadid] * T[getL(d, i + 1, k * blockdim + threadid + 1)];
                reduce_sum(tmp, blockdim);
                __syncthreads();

                if (threadid == 0)
                    Y[i] -= tmp[0];
                tmp[threadid] = 0;

            }
            __syncthreads();
        }
        // usual case
        __syncthreads();
        if (threadid < rmd) {
            tmp[threadid] = Y[q * blockdim + threadid] * T[getL(d, i + 1, q * blockdim + threadid + 1)];

            __syncthreads();
            reduce_sum(tmp, rmd);

            __syncthreads();
            if (threadid == 0)
                Y[i] -= tmp[0];
        }
        tmp[threadid] = 0;
        __syncthreads();
    }
}

__device__ void solve_tsup(float *T, float *Y, int d) {
    // Solve an equation of the form LZ = Y
    // T represents the sup triangular matrix
    // The results is then stored in Z

    extern __shared__ float tmp[];

    int blockdim = blockDim.x;
    int threadid = threadIdx.x;

    for (int i = 1; i <= d; i++) {

        int q = i / blockdim;
        int rmd = i % blockdim;

        for (int k = 0; k < q; k++) {
            __syncthreads();
            if (threadid < blockdim) {

                __syncthreads();
                if (threadIdx.x < blockdim) {
                    tmp[threadid] = Y[d - (k*blockdim+threadid)] * T[getL(d, d - i + 1, d - (k*blockdim+threadid) + 1)];

                    reduce_sum(tmp, blockdim);

                    __syncthreads();
                    if (threadIdx.x == 0)
                        Y[d - i] -= tmp[0];
                }
            }
            tmp[threadid] = 0;
        }

        if (threadid < rmd) {

            __syncthreads();
            if (threadIdx.x < rmd) {
                tmp[threadid] = Y[d - (q*blockdim+threadid)] * T[getL(d, d - i + 1, d - (q*blockdim+threadid) + 1)];

                reduce_sum(tmp, rmd);

                __syncthreads();
                if (threadIdx.x == 0)
                    Y[d - i] -= tmp[0];
            }
            tmp[threadid] = 0;
        }
    }
}

__device__ void invert_diag(float *D, float *Y, int d) {
    int q = d / blockDim.x;
    int rmd = d % blockDim.x;

    for (int k = 0; k < q; k++) {
        if (threadIdx.x < blockDim.x)
            Y[k * blockDim.x + threadIdx.x] /= D[k * blockDim.x + threadIdx.x];
    }

    if (threadIdx.x < rmd)
        Y[q * blockDim.x + threadIdx.x] /= D[q * blockDim.x + threadIdx.x];

}

__device__ void solve_system(float *D, float *T, float *Y, int d) {
    solve_tinf(T, Y, d);
    invert_diag(D, Y, d);
    solve_tsup(T, Y, d);
}

__global__ void solve_batch(int N, int d, float *T, float *Y) {
    int matrix_size = d + d * (d + 1) / 2;
    solve_system(&T[matrix_size * blockIdx.x], &T[matrix_size * blockIdx.x + d], &Y[matrix_size * blockIdx.x], d);
}